#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define blockSize 512
#define real double


template <int bSize> __global__ void redukcja (int N, real* v, real* out)
{
 size_t s = threadIdx.x + blockIdx.x * blockDim.x*2;
 int sID = threadIdx.x;
 size_t i;

 __shared__ real pom[blockSize];
 
 pom[sID] = 0;
 if (s<N/2)
	 pom[sID] = v[s] + v[s + blockDim.x];
 __syncthreads();

 if (bSize >=512) {
 	if (sID<256) pom[sID] += pom[sID + 256]; __syncthreads();
 }
 if (bSize >=256) {
	if (sID<128) pom[sID] += pom[sID + 128]; __syncthreads();
 }
 if (bSize >= 128){
 	if (sID<64) pom[sID] += pom[sID + 64]; __syncthreads();
 }	
 if (sID < 32){
	if (bSize >= 64) pom[sID] += pom[sID + 32];
__syncthreads();	
	if (bSize >= 32) pom[sID] += pom[sID + 16];
__syncthreads();	
	if (bSize >= 16) pom[sID] += pom[sID +  8];
__syncthreads();	
	if (bSize >= 8) pom[sID] += pom[sID +  4];
__syncthreads();	
	if (bSize >= 4) pom[sID] += pom[sID +  2];
__syncthreads();	
	if (bSize >= 2) pom[sID] += pom[sID +  1];
__syncthreads();	
 }	
 if (sID==0) out[blockIdx.x] = pom[0];
}

__global__ void redukcja2 (int N, real* v, real* out)
{
 size_t s = threadIdx.x + blockIdx.x * blockDim.x;
 int sID = threadIdx.x;
 size_t i;

 __shared__ real pom[blockSize];

 pom[sID] = 0;
 if (s<N)
         pom[sID] = v[s];
 __syncthreads();

 for (i=blockDim.x/2; i>0; i>>=1){
        if (sID<i){
                pom[sID] += pom[sID + i];
        }
        __syncthreads();
 }
 if (sID==0) out[blockIdx.x] = pom[0];
}

__global__ void wypelnij (int N, real* v)
{
 size_t s = threadIdx.x + blockIdx.x * blockDim.x;

 if (s<N) {
 	v[s] = sin(s * 2. * M_PI / 10.);
 }	
}

int main ()
{
 size_t N = blockSize * blockSize * blockSize;
 int blocks = (N + blockSize-1) / blockSize;

 float dt_ms;

 hipEvent_t event1, event2;
 hipEventCreate(&event1);
 hipEventCreate(&event2);

 real* v;
 hipMalloc( (void**) &v, N * sizeof(real) );
 real* outV;
 hipMalloc( (void**) &outV, blockSize * blockSize * sizeof(real) );
 real* outVV;
 hipMalloc( (void**) &outVV, blockSize * sizeof(real) );

 real out;
 int i;
 int M = 10;

 wypelnij <<<blocks, blockSize>>> (N, v);


 hipEventRecord(event1, 0);
 	for (i=0; i<M; i++){
 		redukcja<blockSize><<<blocks/2, blockSize>>> (N, v, outV);
		redukcja<blockSize><<<blockSize/2, blockSize>>> (blockSize*blockSize, outV, outVV);
		redukcja2<<<1, blockSize>>> (blockSize, outVV, v);
	}
 hipEventRecord(event2, 0);

 hipEventSynchronize(event1);
 hipEventSynchronize(event2);

 hipEventElapsedTime(&dt_ms, event1, event2);
 hipMemcpy (&out, v, 1 * sizeof(real), hipMemcpyDeviceToHost);
  
 printf ("Czas redukcji: %f ms   wynik; %f\n", dt_ms * 1./M, out);

 return 0;
} 
