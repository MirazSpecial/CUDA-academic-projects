#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define blockSize 512
#define real float


__global__ void redukcja (int N, real* v)
{
 size_t s = threadIdx.x + blockIdx.x * blockDim.x;
 size_t i;

 real p = 0;
 if (s==0){
//	*out = 0;
	for (i=0; i<N; i++)
		p += v[i];
 	v[0] = p;		
 }		
}

__global__ void wypelnij (int N, real* v)
{
 size_t s = threadIdx.x + blockIdx.x * blockDim.x;

 if (s<N) {
 	v[s] = sin(s * 2. * M_PI / 10.);
 }	
}

int main ()
{
 size_t N = blockSize*blockSize*blockSize;
 int blocks = (N + blockSize-1) / blockSize;
 float dt_ms;

 hipEvent_t event1, event2;
 hipEventCreate(&event1);
 hipEventCreate(&event2);

 real* v;
 hipMalloc( (void**) &v, N * sizeof(real) );

 real out;
 int i;
 int M = 10;

 wypelnij <<<blocks, blockSize>>> (N, v);


 hipEventRecord(event1, 0);
 	for (i=0; i<M; i++){
 		redukcja<<<blocks, blockSize>>> (N, v);
	
	}
 hipEventRecord(event2, 0);

 hipEventSynchronize(event1);
 hipEventSynchronize(event2);

 hipEventElapsedTime(&dt_ms, event1, event2);
 hipMemcpy (&out, v, 1 * sizeof(real), hipMemcpyDeviceToHost);
  
 printf ("Czas redukcji: %f ms   wynik; %f\n", dt_ms * 1./M, out);

 return 0;
} 
